#include "hip/hip_runtime.h"
#include "kernel.cu"
#include <iostream>

#define BLOCKSIZE 1024
int main() {
  unsigned char *d_grid_max;
  int deviceId;
  hipDeviceProp_t prop;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, nullptr);
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&prop, deviceId);
  int sm_count = prop.multiProcessorCount;
  int block_per_sm = 0;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&block_per_sm, rng, BLOCKSIZE,
                                                0);
  int block_count = sm_count * block_per_sm;
  hipMallocManaged(&d_grid_max, block_count);
  rng<<<block_count, BLOCKSIZE>>>(d_grid_max, 42);
  // rng<<<block_count, BLOCKSIZE>>>(d_grid_max, time(nullptr));
  hipDeviceSynchronize();
  hipEventRecord(stop, nullptr);
  float t = 0;
  int global_max = d_grid_max[0];
  for (int i = 1; i < block_count; i++) {
    global_max = max(global_max, d_grid_max[i]);
  }
  hipEventElapsedTime(&t, start, stop);
  std::cout << "kernel ran in " << t << "\n";
  std::cout << "Max: " << global_max << '\n';
  hipFree(d_grid_max);
  return 0;
}
