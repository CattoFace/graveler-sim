#include "hip/hip_runtime.h"
#include "kernel.cu"
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define BLOCKSIZE 1024
int main() {
  unsigned char *d_grid_max;
  int deviceId;
  hipDeviceProp_t prop;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&prop, deviceId);
  int sm_count = prop.multiProcessorCount;
  int block_per_sm = 0;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&block_per_sm, rng, BLOCKSIZE,
                                                0);
  int block_count = sm_count * block_per_sm;
  hipMallocManaged(&d_grid_max, block_count);
  int black_box = 0; // to prevent optimizing away the entire loop
  int global_max;
  // warm-up
  for (int i = 0; i < 10; i++) {
    rng<<<block_count, BLOCKSIZE>>>(d_grid_max, time(nullptr));
    global_max = d_grid_max[0];
    for (int i = 1; i < block_count; i++) {
      global_max = max(global_max, d_grid_max[i]);
    }
    black_box += global_max;
    hipDeviceSynchronize();
  }
  hipEventRecord(start, nullptr);
  for (int i = 0; i < 10; i++) {
    rng<<<block_count, BLOCKSIZE>>>(d_grid_max, time(nullptr));
    global_max = d_grid_max[0];
    for (int i = 1; i < block_count; i++) {
      global_max = max(global_max, d_grid_max[i]);
    }
    black_box += global_max;
    hipDeviceSynchronize();
  }
  hipEventRecord(stop, nullptr);
  float t = 0;
  std::cout << "Max: " << global_max << '\n';
  hipEventElapsedTime(&t, start, stop);
  std::cout << "average of 10 runs " << t / 10 << "ms\n";
  hipFree(d_grid_max);
  std::cout << black_box << "\n";
  return 0;
}
